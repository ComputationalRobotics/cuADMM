#include "hip/hip_runtime.h"
/*

    solver.cu

    Main solver, works for any sizes of matrices.
    Uses the sGS-ADMM algorithm to solve an SDP problem.

*/

#include "cuadmm/solver.h"
#include "cuadmm/kernels.h"
#include "cuadmm/projection.h"

#include <algorithm>
#include <stdio.h>

#define SIG_UPDATE_THRESHOLD 500
#define SIG_UPDATE_STAGE_1 50
#define SIG_UPDATE_STAGE_2 100
#define SIG_SCALE 1.05

void SDPSolver::synchronize_gpu0_streams() {
    CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
    CHECK_CUDA( hipStreamSynchronize(this->stream_flex[1].stream) );
    CHECK_CUDA( hipStreamSynchronize(this->stream_flex[2].stream) );
}

void SDPSolver::init(
    int eig_stream_num_per_gpu,
    int cpu_eig_thread_num,
    int vec_len, int con_num,
    int* cpu_At_csc_col_ptrs, int* cpu_At_csc_row_ids, double* cpu_At_csc_vals, int At_nnz,
    int* cpu_b_indices, double* cpu_b_vals, int b_nnz,
    int* cpu_C_indices, double* cpu_C_vals, int C_nnz,
    char* cpu_blk_types, int* cpu_blk_sizes,
    int mat_num,
    double* cpu_X_vals,
    double* cpu_y_vals,
    double* cpu_S_vals,
    double sig
) {
    // start record time
    this->total_time = 0.0;
    hipEventCreate(&this->start);
    hipEventCreate(&this->stop);
    hipEventRecord(this->start);

    // prepare streams for copy data
    /*
    we create three flexible streams per GPU, corresponding to copy mom_mat, mom_W, mom_info
    they can also be used to parallelize kernel launches and cuda toolkit calls
    */
    this->stream_flex = std::vector<DeviceStream>(3);
    for (int stream_id = 0; stream_id < 3; stream_id++) {
        this->stream_flex[stream_id].set_gpu_id(GPU0);
        this->stream_flex[stream_id].activate();
    }

    // create handles for cuSPARSE and cuBLAS
    this->cusparseH.set_gpu_id(GPU0);
    this->cusparseH.activate();
    this->cublasH.set_gpu_id(GPU0);
    this->cublasH.activate();

    /* Initialize the A matrix */
    this->vec_len = vec_len;
    this->con_num = con_num;
    this->At_csc.allocate(GPU0, vec_len, con_num, At_nnz);
    this->At_csr.allocate(GPU0, vec_len, con_num, At_nnz);
    this->A_csr.allocate(GPU0, con_num, vec_len, At_nnz);
    // first stream for col_ptrs
    CHECK_CUDA( hipMemcpyAsync(this->At_csc.col_ptrs, cpu_At_csc_col_ptrs, sizeof(int) * (con_num + 1), H2D, this->stream_flex[0].stream) );
    // second stream for row_ids
    CHECK_CUDA( hipMemcpyAsync(this->At_csc.row_ids, cpu_At_csc_row_ids, sizeof(int) * At_nnz, H2D, this->stream_flex[1].stream) );
    // third stream for vals
    CHECK_CUDA( hipMemcpyAsync(this->At_csc.vals, cpu_At_csc_vals, sizeof(double) * At_nnz, H2D, this->stream_flex[2].stream) );
    // wait for the streams to finish
    this->synchronize_gpu0_streams();

    // compute the norm of A
    this->normA.allocate(GPU0, con_num);
    get_normA(this->At_csc, this->normA);

    /* convert the At matrix from CSC to CSR format */
    this->CSCtoCSR_At2A_buffer_size = CSC_to_CSR_get_buffersize_cusparse(this->cusparseH, this->At_csc, this->At_csr);
    this->CSCtoCSR_At2A_buffer.allocate(GPU0, CSCtoCSR_At2A_buffer_size, true);
    CSC_to_CSR_cusparse(this->cusparseH, this->At_csc, this->At_csr, this->CSCtoCSR_At2A_buffer);
    CHECK_CUDA( hipMemcpyAsync(this->A_csr.row_ptrs ,this->At_csc.col_ptrs, sizeof(int) * (con_num + 1), D2D, this->stream_flex[0].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->A_csr.col_ids ,this->At_csc.row_ids, sizeof(int) * At_nnz, D2D, this->stream_flex[1].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->A_csr.vals ,this->At_csc.vals, sizeof(double) * At_nnz, D2D, this->stream_flex[2].stream) );

    /* Initialize the AAt solver on CPU */
    this->cpu_AAt_solver.get_A(
        this->At_csr.row_ptrs, this->At_csr.col_ids, this->At_csr.vals,
        this->At_csr.col_size, this->At_csr.row_size, this->At_csr.nnz,
        true, 1e-15
    );
    this->cpu_AAt_solver.factorize();
    // retrieve permutation of the L factor
    this->perm.allocate(GPU0, con_num);
    CHECK_CUDA( hipMemcpyAsync(this->perm.vals, this->cpu_AAt_solver.chol_fac_L->Perm, sizeof(int) * con_num, H2D, this->stream_flex[0].stream) );
    // allocate memory of right-hand side vector
    this->rhsy.allocate(GPU0, con_num);
    this->rhsy_perm.allocate(GPU0, con_num);
    this->y_perm.allocate(GPU0, con_num);
    // compute inverse permutation
    std::vector<int> perm_tmp(con_num, 0);
    std::vector<int> perm_inv_tmp;
    memcpy(perm_tmp.data(), this->cpu_AAt_solver.chol_fac_L->Perm, sizeof(int) * con_num);
    this->perm_inv.allocate(GPU0, con_num);
    get_inverse_permutation(perm_inv_tmp, perm_tmp);
    CHECK_CUDA( hipMemcpyAsync(this->perm_inv.vals, perm_inv_tmp.data(), sizeof(int) * con_num, H2D, this->stream_flex[1].stream) );

    /* Initialize b, C, X, y, S, sig on GPU */
    this->b.allocate(GPU0, con_num, b_nnz);
    this->C.allocate(GPU0, vec_len, C_nnz);
    this->X.allocate(GPU0, vec_len);
    this->y.allocate(GPU0, con_num);
    this->S.allocate(GPU0, vec_len);
    CHECK_CUDA( hipMemcpyAsync(this->b.indices, cpu_b_indices, sizeof(int) * b_nnz, H2D, this->stream_flex[0].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->b.vals, cpu_b_vals, sizeof(double) * b_nnz, H2D, this->stream_flex[1].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->C.indices, cpu_C_indices, sizeof(int) * C_nnz, H2D, this->stream_flex[2].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->C.vals, cpu_C_vals, sizeof(double) * C_nnz, H2D, this->stream_flex[0].stream) );

    // copy X, y, and S from CPU to GPU
    // if the input is nullptr (no warm start), we will set them to 0
    if (cpu_X_vals != nullptr) {
        // copy
        CHECK_CUDA( hipMemcpyAsync(this->X.vals, cpu_X_vals, sizeof(double) * vec_len, H2D, this->stream_flex[1].stream) );
    } else {
        // set to 0
        CHECK_CUDA( hipMemsetAsync(this->X.vals, 0, sizeof(double) * vec_len, this->stream_flex[1].stream) );
    }
    if (cpu_y_vals != nullptr) {
        CHECK_CUDA( hipMemcpyAsync(this->y.vals, cpu_y_vals, sizeof(double) * con_num, H2D, this->stream_flex[2].stream) );
    } else {
        CHECK_CUDA( hipMemsetAsync(this->y.vals, 0, sizeof(double) * con_num, this->stream_flex[2].stream) );
    }
    if (cpu_S_vals != nullptr) {
        CHECK_CUDA( hipMemcpyAsync(this->S.vals, cpu_S_vals, sizeof(double) * vec_len, H2D, this->stream_flex[0].stream) );
    } else {
        CHECK_CUDA( hipMemsetAsync(this->S.vals, 0, sizeof(double) * vec_len, this->stream_flex[0].stream) );
    }
    this->sig = sig;

    /* Initialize blk and maps */
    // copy blk values and analyze it to retrieve the block sizes and numbers
    HostDenseVector<int> host_blk_sizes(mat_num);
    memcpy(host_blk_sizes.vals, cpu_blk_sizes, sizeof(int) * mat_num);
    analyze_blk(cpu_blk_types, host_blk_sizes, this->psd_blk_sizes, this->psd_blk_nums);
    this->sizes.init(this->psd_blk_sizes, this->psd_blk_nums);

    /* Compute the maps for vectorization of matrices */
    // compute on CPU
    std::vector<int> map_B_tmp;  // |
    std::vector<int> map_M1_tmp; // |- CPU version
    std::vector<int> map_M2_tmp; // |
    get_maps(cpu_blk_types, host_blk_sizes, this->vec_len, map_B_tmp, map_M1_tmp, map_M2_tmp, this->sizes);

    // copy to GPU
    this->map_B.allocate(GPU0, vec_len);  // |
    this->map_M1.allocate(GPU0, vec_len); // |- GPU version
    this->map_M2.allocate(GPU0, vec_len); // |
    CHECK_CUDA( hipMemcpyAsync(this->map_B.vals, map_B_tmp.data(), sizeof(int) * vec_len, H2D, this->stream_flex[0].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->map_M1.vals, map_M1_tmp.data(), sizeof(int) * vec_len, H2D, this->stream_flex[1].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->map_M2.vals, map_M2_tmp.data(), sizeof(int) * vec_len, H2D, this->stream_flex[2].stream) );

    /* Scale (A is already scaled) */
    // move b and C to GPU
    this->borg.allocate(GPU0, this->con_num, this->b.nnz);
    this->Corg.allocate(GPU0, this->vec_len, this->C.nnz);
    CHECK_CUDA( hipMemcpyAsync(this->borg.indices, this->b.indices, sizeof(int) * this->b.nnz, D2D, this->stream_flex[0].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->borg.vals, this->b.vals, sizeof(double) * this->b.nnz, D2D, this->stream_flex[1].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->Corg.indices, this->C.indices, sizeof(int) * this->C.nnz, D2D, this->stream_flex[2].stream) );
    CHECK_CUDA( hipMemcpyAsync(this->Corg.vals, this->C.vals, sizeof(double) * this->C.nnz, D2D, this->stream_flex[0].stream) );
    this->synchronize_gpu0_streams();
    // compute the norms of b and C
    this->norm_borg = 1 + this->borg.get_norm(this->cublasH);
    this->norm_Corg = 1 + this->Corg.get_norm(this->cublasH);

    // scale b and C by normA
    sparse_vector_div_dense_vector(this->b, this->normA);
    dense_vector_mul_dense_vector(this->y, this->normA);
    // divide b, C, X, y, and S by the corresponding norms
    this->bscale = 1 + this->b.get_norm(this->cublasH);
    this->Cscale = 1 + this->C.get_norm(this->cublasH);
    this->objscale = this->bscale * this->Cscale;
    sparse_vector_div_scalar(this->b, this->bscale);
    sparse_vector_div_scalar(this->C, this->Cscale);
    dense_vector_div_scalar(this->X, this->bscale);
    dense_vector_div_scalar(this->S, this->Cscale);
    dense_vector_div_scalar(this->y, this->Cscale);

    /* Initialize KKT residuals */
    // simple allocations
    this->Aty.allocate(GPU0, this->vec_len);
    this->Rp.allocate(GPU0, this->con_num);
    this->SmC.allocate(GPU0, this->vec_len);
    this->Rd.allocate(GPU0, this->vec_len);
    this->Rporg.allocate(GPU0, this->con_num);
    this->Rdorg.allocate(GPU0, this->vec_len);

    // retrieve buffer sizes and allocate
    this->SpMV_Aty_buffer_size = SpMV_get_buffersize_cusparse(this->cusparseH, this->At_csr, this->y, this->Aty, 1.0, 0.0);
    this->SpMV_Aty_buffer.allocate(GPU0, this->SpMV_Aty_buffer_size, true);
    SpMV_cusparse(this->cusparseH, this->At_csr, this->y, this->Aty, 1.0, 0.0, this->SpMV_Aty_buffer);
    this->SpMV_AX_buffer_size = SpMV_get_buffersize_cusparse(this->cusparseH, this->A_csr, this->X, this->Rp, -1.0, 0.0);
    this->SpMV_AX_buffer.allocate(GPU0, this->SpMV_AX_buffer_size, true);
    SpMV_cusparse(this->cusparseH, this->A_csr, this->X, this->Rp, -1.0, 0.0, this->SpMV_AX_buffer);

    //
    axpby_cusparse(this->cusparseH, this->b, this->Rp, 1.0, 1.0);
    CHECK_CUDA( hipMemcpy(this->SmC.vals, this->S.vals, sizeof(double) * this->vec_len, D2D) );
    axpby_cusparse(this->cusparseH, this->C, this->SmC, -1.0, 1.0);
    dense_vector_add_dense_vector(this->Rd, this->Aty, this->SmC);
    dense_vector_mul_dense_vector_mul_scalar(this->Rporg, this->normA, this->Rp, this->bscale);
    dense_vector_mul_scalar(this->Rdorg, this->Rd, this->Cscale);

    // compute initial residuals
    this->errRp = this->Rporg.get_norm(this->cublasH) / this->norm_borg;
    this->errRd = this->Rdorg.get_norm(this->cublasH) / this->norm_Corg;
    this->maxfeas = max(this->errRp, this->errRd);
    this->SpVV_CtX_buffer_size = SparseVV_get_buffersize_cusparse(this->cusparseH, this->C, this->X);
    this->SpVV_CtX_buffer.allocate(GPU0, this->SpVV_CtX_buffer_size, true);
    this->pobj = SparseVV_cusparse(this->cusparseH, this->C, this->X, this->SpVV_CtX_buffer) * this->objscale;
    this->SpVV_bty_buffer_size = SparseVV_get_buffersize_cusparse(this->cusparseH, this->b, this->y);
    this->SpVV_bty_buffer.allocate(GPU0, this->SpVV_bty_buffer_size, true);
    this->dobj = SparseVV_cusparse(this->cusparseH, this->b, this->y, this->SpVV_bty_buffer) * this->objscale;
    this->relgap = abs(this->pobj - this->dobj) / (1 + abs(this->pobj) + abs(this->dobj));

    /* Eigen decomposition for large matrices */
    // allocate GPU0 memory for large matrices
    this->large_mat.allocate(GPU0, this->sizes.total_large_mat_size);
    this->large_W.allocate(GPU0, this->sizes.sum_large_mat_size);
    this->large_info.allocate(GPU0, this->sizes.large_mat_num);

    // if the decomposition is on GPU, use cuSOLVER (cf cusolver.h)
    this->eig_stream_num_per_gpu = eig_stream_num_per_gpu;

    // streams and handles for eigen decomposition
    this->eig_stream_arr = std::vector<DeviceStream>(this->eig_stream_num_per_gpu);
    this->cusolverH_eig_large_arr = std::vector<DeviceSolverDnHandle>(this->eig_stream_num_per_gpu);
    for (int stream_id = 0; stream_id < this->eig_stream_num_per_gpu; stream_id++) {
        // ininitialize and activate the streams and handles
        this->eig_stream_arr[stream_id].set_gpu_id(GPU0);
        this->eig_stream_arr[stream_id].activate();
        this->cusolverH_eig_large_arr[stream_id].set_gpu_id(GPU0);
        this->cusolverH_eig_large_arr[stream_id].activate(this->eig_stream_arr[stream_id]);
    }
    
    // compute the buffer sizes of the large matrices eig decomposition
    this->eig_large_buffer_size.assign(this->sizes.large_mat_sizes.size(), 0);
    // this->eig_large_buffer.reserve(this->sizes.large_mat_sizes.size());
    this->cpu_eig_large_buffer_size.assign(this->sizes.large_mat_sizes.size(), 0);
    // this->cpu_eig_large_buffer.reserve(this->sizes.large_mat_sizes.size());

    // TODO: comment out this part
    this->sizes.large_buffer_start_indices.push_back(0);
    this->sizes.large_cpu_buffer_start_indices.push_back(0);
    int total_eig_large_buffer_size = 0;
    int total_cpu_eig_large_buffer_size = 0;
    for (int i = 0; i < this->sizes.large_mat_sizes.size(); i++) {
        single_eig_get_buffersize_cusolver(
            this->cusolverH_eig_large_arr[i % this->eig_stream_num_per_gpu], eig_param_single, this->large_mat, this->large_W,
            this->sizes.large_mat_sizes[i],
            &this->eig_large_buffer_size[i],
            &this->cpu_eig_large_buffer_size[i],
            this->sizes.large_mat_offset(i, 0), this->sizes.large_W_offset(i, 0)
        ); // buffer size per large matrix of a given size

        // we need to multiply the buffer size by the number of matrices of this size
        total_eig_large_buffer_size += this->eig_large_buffer_size[i] * this->sizes.large_mat_nums[i];
        total_cpu_eig_large_buffer_size += this->cpu_eig_large_buffer_size[i] * this->sizes.large_mat_nums[i];

        this->sizes.large_buffer_start_indices.push_back(
            this->sizes.large_buffer_start_indices[i] + this->sizes.large_mat_nums[i] * this->eig_large_buffer_size[i]
        );
        this->sizes.large_cpu_buffer_start_indices.push_back(
            this->sizes.large_cpu_buffer_start_indices[i] + this->sizes.large_mat_nums[i] * this->cpu_eig_large_buffer_size[i]
        );
    }

    // allocate memory for the two buffers, host and device
    this->eig_large_buffer.allocate(GPU0, total_eig_large_buffer_size, true);
    this->cpu_eig_large_buffer.allocate(total_cpu_eig_large_buffer_size, true);

    /* Eigenvalue decomposition for small matrices */
    this->cusolverH_eig_small.set_gpu_id(GPU0);
    this->cusolverH_eig_small.activate();
    this->small_mat.allocate(GPU0, this->sizes.total_small_mat_size);
    this->small_W.allocate(GPU0, this->sizes.sum_small_mat_size);
    this->small_info.allocate(GPU0, this->sizes.small_mat_num);
    this->eig_small_buffer_size.reserve(this->sizes.small_mat_sizes.size());

    this->sizes.small_buffer_start_indices.push_back(0);
    for (int i = 0; i < this->sizes.small_mat_sizes.size(); i++) {
        this->eig_small_buffer_size.push_back(
            batch_eig_get_buffersize_cusolver(
                this->cusolverH_eig_small, this->eig_param_batch,
                this->small_mat, this->small_W,
                this->sizes.small_mat_sizes[i], this->sizes.small_mat_nums[i],
                this->sizes.small_mat_offset(i), this->sizes.small_W_offset(i)
            )
        );

        this->sizes.small_buffer_start_indices.push_back(
            this->sizes.small_buffer_start_indices[i] + this->eig_small_buffer_size[i]
        );
    }
    
    CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
    // we do not need to multiply the buffer size by the number of matrices,
    // since it is already done in the function
    this->eig_small_buffer.allocate(GPU0, this->sizes.small_buffer_start_indices.back(), true);

    /* For the computation of y, X, S */
    this->large_mat_tmp.allocate(GPU0, this->sizes.total_large_mat_size);
    this->small_mat_tmp.allocate(GPU0, this->sizes.total_small_mat_size);
    this->large_mat_P.allocate(GPU0, this->sizes.total_large_mat_size); // TODO: remove this
    this->small_mat_P.allocate(GPU0, this->sizes.total_small_mat_size);
    this->Rd1.allocate(GPU0, this->vec_len);
    this->Xb.allocate(GPU0, this->vec_len);

    /* others */
    this->prim_win = 0;
    this->dual_win = 0;
    this->ratioconst = 1e0;
    this->sigmax = 1e3;
    this->sigmin = 1e-3;

    /* Main elements for the sGS-ADMM algorithm */
    this->Xproj.allocate(GPU0, this->vec_len);
    this->Xdiff.allocate(GPU0, this->vec_len);
    this->switch_admm = (int) 5e4;
    this->sig_update_threshold = SIG_UPDATE_THRESHOLD;
    this->sig_update_stage_1 = SIG_UPDATE_STAGE_1;
    this->sig_update_stage_2 = SIG_UPDATE_STAGE_2;
    this->sigscale = SIG_SCALE;
    this->X_best.allocate(GPU0, this->vec_len);
    this->y_best.allocate(GPU0, this->con_num);
    this->S_best.allocate(GPU0, this->vec_len);

    return;
}

// Solves the SDP problem using the sGS-ADMM algorithm.
//
// Args:
// - max_iter: maximum number of iterations
// - stop_tol: stopping tolerance for KKT residual
// - sig_update_threshold:
// - sig_update_stage_1:
// - sig_update_stage_2:
// - switch_admm:
// - sigscale:
// - if_first: if this is the first call to solve() (optional)
void SDPSolver::solve(
    int max_iter, double stop_tol,
    int sig_update_threshold,
    int sig_update_stage_1,
    int sig_update_stage_2,
    int switch_admm,
    double sigscale,
    bool if_first
) {
    // save parameters
    this->sig_update_threshold = sig_update_threshold;
    this->sig_update_stage_1 = sig_update_stage_1;
    this->sig_update_stage_2 = sig_update_stage_2;
    this->switch_admm = switch_admm;
    this->sigscale = sigscale;

    // declare variables
    bool breakyes = false;   // for breaking out of the loop
    std::string final_msg;   // output message

    this->info_iter_num = 0; // iteration number

    /* Start the solver */
    printf("\n -------------------------------------------------------------------------------");
    printf("\n                                    cuADMM");
    printf("\n -------------------------------------------------------------------------------");
    printf("\n norm of C = %2.1e, norm of b = %2.1e\n", norm_Corg, norm_borg);
    float milliseconds;
    float seconds;

    if (!if_first) {
        // we suppose that for the second call, new X, y, S, sig are passed, but they are unscaled

        // scale X, y, S
        dense_vector_mul_dense_vector(this->y, this->normA);
        dense_vector_div_scalar(this->X, this->bscale);
        dense_vector_div_scalar(this->S, this->Cscale);
        dense_vector_div_scalar(this->y, this->Cscale);

        // SmC <-- S
        CHECK_CUDA( hipMemcpy(this->SmC.vals, this->S.vals, sizeof(double) * this->vec_len, D2D) );
        // hence Smc = S

        // SmC <-- -1.0 * C + 1.0 * SmC
        axpby_cusparse(this->cusparseH, this->C, this->SmC, -1.0, 1.0);
        // hence SmC = S - C

        // Rp <-- -1.0 * A * X + 0.0 * Rp
        SpMV_cusparse(this->cusparseH, this->A_csr, this->X, this->Rp, -1.0, 0.0, this->SpMV_AX_buffer);
        // hence Rp = - A X

        // Rp <-- 1.0 * b + 1.0 * Rp
        axpby_cusparse(this->cusparseH, this->b, this->Rp, 1.0, 1.0);
        // hence Rp = b - A X
    }

    std::cout << std::endl << "  it. | p infeas d infeas | primal obj.   dual obj. rel. gap |  time |   sigma | " << std::endl;
    std::cout << " -------------------------------------------------------------------------------" << std::endl;

    // for each iteration of the main solver
    for (int iter = 1; iter <= max_iter + 1; iter++) {
        /*
            Step 0: Check if terminal conditions hold and log information
        */
        if (max(this->maxfeas, this->relgap) < stop_tol ) {
            // stop if the stopping criterion is met
            breakyes = true;
            final_msg = "Solver ended: converged.";
        }
        if (iter > max_iter) {
            // stop if the maximum number of iterations is reached
            breakyes = true;
            final_msg = "Solver ended: maximum iteration reached";
        }
        if (
            true || // TODO: remove
            ( breakyes == true ) ||
            ( (iter <= 200) && ((iter % 50) == 1) ) ||
            ( (iter > 200) && ((iter % 100) == 1) )
        ) {
            // print the iteration number and the residuals
            hipEventRecord(this->stop);
            hipEventSynchronize(this->stop);
            hipEventElapsedTime(&milliseconds, this->start, this->stop);
            seconds = milliseconds / 1000;
            printf(
                " %4d | %3.2e %3.2e | %- 5.4e %- 5.4e %3.2e | %5.1f | %2.1e |",
                iter-1, this->errRp, this->errRd, this->pobj, this->dobj, this->relgap, seconds, this->sig
            );
            std::cout << std::endl;
        }
        if (breakyes > 0) {
            // print the final message
            printf("\n -------------------------------------------------------------------------------\n\n");
            std::cout << final_msg << std::endl;
            printf(
                "\n primal infeasibility = %2.1e \n dual   infeasibility = %2.1e \n relative gap         = %2.1e",
                this->errRp, this->errRd, this->relgap
            );
            printf(
                "\n primal objective = %- 9.8e \n dual   objective = %- 9.8e",
                this->pobj, this->dobj
            );
            printf(
                "\n\n time per iteration = %2.4fs \n total time         = %2.1fs",
                seconds/iter, seconds
            );
            printf("\n -------------------------------------------------------------------------------\n\n");

            hipEventRecord(this->stop);
            hipEventSynchronize(this->stop);
            hipEventElapsedTime(&milliseconds, this->start, this->stop);
            this->total_time = milliseconds / 1000;
        }

        /*
            Step 1: Compute
                        r_s^{k+1/2} = 1/sigma b - A(X/sigma + S^k - C)
                                             and
                               y^{k+1/2} = (AA^T)^{-1} r_s^{k+1/2}
        */

        /* r_s^{k+1/2} = b/sigma - A(X/sigma + S - C) */
        // rhsy <-- -1.0 * A * SmC + 0.0 * rhsy
        SpMV_cusparse(this->cusparseH, this->A_csr, this->SmC, this->rhsy, -1.0, 0.0, this->SpMV_AX_buffer);
        // hence rhsy = - A S

        // rhsy <-- 1/sig * Rp + rhsy
        axpy_cublas(this->cublasH, this->Rp, this->rhsy, 1/this->sig);
        // hence rhsy = 1/sig * Rp - A S

        /* y^{k+1/2} = (AA^T)^{-1} r_s^{k+1/2} */
        // y <-- linsys(rhsy)
        perform_permutation(this->rhsy_perm, this->rhsy, this->perm_inv);
        CHECK_CUDA( hipDeviceSynchronize() );
        CHECK_CUDA( hipMemcpyAsync(
            this->cpu_AAt_solver.chol_dn_rhs->x, this->rhsy_perm.vals,
            sizeof(double) * this->con_num, D2H, this->stream_flex[0].stream
        ) );
        CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
        this->cpu_AAt_solver.solve();
        CHECK_CUDA( hipMemcpyAsync(
            this->y_perm.vals, this->cpu_AAt_solver.chol_dn_res->x,
            sizeof(double) * this->con_num, H2D, this->stream_flex[0].stream
        ) );
        CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
        perform_permutation(this->y, this->y_perm, this->perm);
        // hence y = (AA^T)^{-1} r_s^{k+1/2}


        /*
            Step 2: Compute the optimization variables :

                    X_b^{k+1} = X^k + sigma(A^T y^{k+1/2} - C)
                                         and
                    S^{k+1} = 1/sigma (Pi(X_b^{k+1}) - X_b^{k+1})
        */

        /* Compute X^{k+1} */
        // Aty <-- 1.0 * At * y + 0.0 * Aty
        SpMV_cusparse(this->cusparseH, this->At_csr, this->y, this->Aty, 1.0, 0.0, this->SpMV_Aty_buffer);
        // hence Aty = A^T y^{k+1/2}

        // Rd1 <-- Aty
        CHECK_CUDA( hipMemcpy(this->Rd1.vals, this->Aty.vals, sizeof(double) * this->vec_len, D2D) );
        // Rd1 <-- (-1.0) * C + 1.0 * Rd1
        axpby_cusparse(this->cusparseH, this->C, this->Rd1, -1.0, 1.0);
        // hence Rd1 = A^T y^{k+1/2} - C

        double norm_rhsy = this->rhsy.get_norm(this->cublasH);
        double norm_y = this->y.get_norm(this->cublasH);

        // Xb <-- X + sig * Rd1
        dense_vector_plus_dense_vector_mul_scalar(this->Xb, this->X, this->Rd1, this->sig);
        // hence Xb = X^k + sig * (A^T y^{k+1/2} - C) = X^{k+1}


        /* Compute Pi(X^{k+1}) (this is long) */

        // first, we convert Xb back to matrices (large and small)
        vector_to_matrices(this->Xb, this->large_mat, this->small_mat, this->map_B, this->map_M1, this->map_M2);

        // we perform the GPU decomposition of large matrices
        // for each large matrix on this GPU, compute the eig decomposition
        // int stream_id;
        // int counter = 0; // serves as a stream id and as an info offset

        // TODO: move the cuBLAS handle creation outside the loop
        hipblasHandle_t cublasH;
        CHECK_CUBLAS( hipblasCreate(&cublasH) );
        CHECK_CUBLAS( hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH) );

        for (int i = 0; i < this->sizes.large_mat_sizes.size(); i++) {
            for (int j = 0; j < this->sizes.large_mat_nums[i]; j++) {
                // stream_id = counter % this->eig_stream_num_per_gpu;

                // simply calls the cuSOLVER wrapper
                // single_eig_cusolver(
                //     this->cusolverH_eig_large_arr[stream_id], eig_param_single,
                //     this->large_mat, this->large_W,
                //     this->eig_large_buffer, this->cpu_eig_large_buffer, this->large_info,
                //     this->sizes.large_mat_sizes[i],
                //     this->eig_large_buffer_size[i], this->cpu_eig_large_buffer_size[i],
                //     this->sizes.large_mat_offset(i, j), this->sizes.large_W_offset(i, j),
                //     this->sizes.large_buffer_offset(i, j, this->eig_large_buffer_size),
                //     this->sizes.large_cpu_buffer_offset(i, j, this->eig_large_buffer_size),
                //     counter
                // );

                // use the custom routine
                projection_TF16(
                    cublasH,
                    this->large_mat,
                    this->sizes.large_mat_sizes[i],
                    this->sizes.large_mat_offset(i, j)
                );

                // counter++;
            }
        }
        hipblasDestroy(cublasH);
        // for each stream, synchronize
        for (int stream_id = 0; stream_id < this->eig_stream_num_per_gpu; stream_id++) {
            CHECK_CUDA( hipStreamSynchronize(this->eig_stream_arr[stream_id].stream) );
        }

        // we perform an ADMM switch
        if (breakyes) {
            if (iter > this->switch_admm) {
                CHECK_CUDA( hipMemcpyAsync(this->X.vals, this->X_best.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[0].stream) );
                CHECK_CUDA( hipMemcpyAsync(this->y.vals, this->y_best.vals, sizeof(double) * this->con_num, D2D, this->stream_flex[1].stream) );
                CHECK_CUDA( hipMemcpyAsync(this->S.vals, this->S_best.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[2].stream) );
                this->synchronize_gpu0_streams();
                printf("best max KKT residual after switch  = %2.1e \n", this->best_KKT);
            }
            break;
        }

        // we call cuSOLVER for the batch eig decomposition of small matrices
        int info_offset = 0;
        for (int i = 0; i < this->sizes.small_mat_sizes.size(); i++) {
            batch_eig_cusolver(
                this->cusolverH_eig_small, this->eig_param_batch,
                this->small_mat, this->small_W,
                this->eig_small_buffer, this->small_info,
                this->sizes.small_mat_sizes[i], this->sizes.small_mat_nums[i],
                this->eig_small_buffer_size[i],
                this->sizes.small_mat_offset(i), this->sizes.small_W_offset(i),
                this->sizes.small_buffer_offset(i, this->eig_small_buffer_size),
                info_offset
            );
            info_offset += this->sizes.small_mat_nums[i];
        }

        // max_dense_vector_zero(this->large_W);
        max_dense_vector_zero(this->small_W);

        // int stream_id;
        // multiply the large matrices by their eigenvalues
        // for (int i = 0; i < this->sizes.large_mat_sizes.size(); i++) {
        //     // stream_id = i % this->eig_stream_num_per_gpu;
        //     dense_matrix_mul_diag_batch(
        //         large_mat_tmp, this->large_mat, this->large_W,
        //         this->sizes.large_mat_sizes[i], this->sizes.large_mat_nums[i],
        //         this->sizes.large_mat_offset(i, 0), this->sizes.large_W_offset(i, 0)//,
        //         // this->eig_stream_arr[stream_id].stream
        //     );
        // }

        // TODO: use multiple streams
        // multiply the small matrices by their eigenvalues
        for (int i = 0; i < this->sizes.small_mat_sizes.size(); i++) {
            // stream_id = (this->sizes.large_mat_sizes.size() + i) % this->eig_stream_num_per_gpu;
            dense_matrix_mul_diag_batch(
                small_mat_tmp, this->small_mat, this->small_W,
                this->sizes.small_mat_sizes[i], this->sizes.small_mat_nums[i],
                this->sizes.small_mat_offset(i), this->sizes.small_W_offset(i)//,
                // this->eig_stream_arr[stream_id].stream
            );
        }

        // synchronize the streams
        // for (int stream_id = 0; stream_id < this->eig_stream_num_per_gpu; stream_id++) {
        //     CHECK_CUDA( hipStreamSynchronize(this->eig_stream_arr[stream_id].stream) );
        // }


        // for (int i = 0; i < this->sizes.large_mat_sizes.size(); i++) {
        //     dense_matrix_mul_trans_batch(
        //         this->cublasH,
        //         this->large_mat_P, this->large_mat_tmp, this->large_mat,
        //         this->sizes.large_mat_sizes[i], this->sizes.large_mat_nums[i],
        //         this->sizes.large_mat_offset(i, 0)
        //     );
        // }

        // TODO: use multiple cuBLAS handles
        for (int i = 0; i < this->sizes.small_mat_sizes.size(); i++) {
            dense_matrix_mul_trans_batch(
                this->cublasH,
                this->small_mat_P, this->small_mat_tmp, this->small_mat,
                this->sizes.small_mat_sizes[i], this->sizes.small_mat_nums[i],
                this->sizes.small_mat_offset(i)
            );
        }

        // we copy Xb to Xproj since free variables are not modified
        // TODO: only copy the free variables
        CHECK_CUDA( hipMemcpy(this->Xproj.vals, this->Xb.vals, sizeof(double) * this->vec_len, D2D) );

        // convert the matrices back to vectorized format
        // matrices_to_vector(this->Xproj, this->large_mat_P, this->small_mat_P, this->map_B, this->map_M1, this->map_M2);
        matrices_to_vector(this->Xproj, this->large_mat, this->small_mat_P, this->map_B, this->map_M1, this->map_M2);

        /* Finish the computation of S^{k+1} */

        // Xdiff <-- 1.0 * Xproj + (-1.0) * X
        dense_vector_add_dense_vector(this->Xdiff, this->Xproj, this->X, 1.0, -1.0);
        // hence Xdiff = Pi(X^{k+1}) - X^k

        // S <-- 1/sig * Xdiff + (-1.0) * Rd1
        dense_vector_add_dense_vector(this->S, this->Xdiff, this->Rd1, 1/this->sig, -1.0);
        // hence S = 1/sig * (Pi(X^{k+1}) - X^k) - (A^T y^{k+1/2} - C)
        // which is S = 1/sig * (Pi(X^{k+1}) - X^{k+1})



        /*
            Step 3: Compute:
                        r_s^{k+1} = 1/sigma b - A(X^k/sigma + S^{k+1} - C)
                                              and
                                y^{k+1} = (AA^T)^{-1} r_s^{k+1}
        */

        /* Compute r_s^{k+1} */

        // SmC <-- S
        CHECK_CUDA( hipMemcpy(this->SmC.vals, this->S.vals, sizeof(double) * this->vec_len, D2D) );
        // SmC <-- -1.0 * C + 1.0 * SmC
        axpby_cusparse(this->cusparseH, this->C, this->SmC, -1.0, 1.0);
        // hence SmC = S^{k+1} - C


        /* Compute y^{k+1} */
        // If the number of iterations goes large but sGS-ADMM still fail to converge,
        // switch to ordinary ADMM
        if (iter == this->switch_admm) {
            std::cout << " switching to normal ADMM!" << std::endl;
            this->sig_update_stage_2 = this->sig_update_stage_2 / 2;
            this->sigscale = this->sigscale * 1.23;
            this->sgs_KKT = max(this->maxfeas, this->relgap);
            this->best_KKT = this->sgs_KKT;
            CHECK_CUDA( hipMemcpyAsync(this->X_best.vals, this->X.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[0].stream) );
            CHECK_CUDA( hipMemcpyAsync(this->y_best.vals, this->y.vals, sizeof(double) * this->con_num, D2D, this->stream_flex[1].stream) );
            CHECK_CUDA( hipMemcpyAsync(this->S_best.vals, this->S.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[2].stream) );
        }

        // when before the switch, perform the special sGS-ADMM step
        if (iter < this->switch_admm) {
            // rhsy <-- -1.0 * A * SmC + 0.0 * rhsy
            SpMV_cusparse(this->cusparseH, this->A_csr, this->SmC, this->rhsy, -1.0, 0.0, this->SpMV_AX_buffer);
            // hence rhsy = - A(S - C)

            // rhsy <-- 1/sig * Rp + rhsy
            axpy_cublas(this->cublasH, this->Rp, this->rhsy, 1/this->sig);
            // hence rhsy = 1/sigma Rp - A(S - C) = 1/sigma (b - A(X^k)) - A(S - C)
            // hence rhsy = 1/sigma b - A(X^k /sigma + S^{k+1} - C)

            // y <-- linsys(rhsy)
            perform_permutation(this->rhsy_perm, this->rhsy, this->perm_inv);
            CHECK_CUDA( hipDeviceSynchronize() );
            CHECK_CUDA( hipMemcpyAsync(
                this->cpu_AAt_solver.chol_dn_rhs->x, this->rhsy_perm.vals,
                sizeof(double) * this->con_num, D2H, this->stream_flex[0].stream
            ) );
            CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
            this->cpu_AAt_solver.solve();
            CHECK_CUDA( hipMemcpyAsync(
                this->y_perm.vals, this->cpu_AAt_solver.chol_dn_res->x,
                sizeof(double) * this->con_num, H2D, this->stream_flex[0].stream
            ) );
            CHECK_CUDA( hipStreamSynchronize(this->stream_flex[0].stream) );
            perform_permutation(this->y, this->y_perm, this->perm);
            // hence y = (AA^T)^{-1} r_s^{k+1}

            // Aty <-- 1.0 * At * y + 0.0 * Aty
            SpMV_cusparse(this->cusparseH, this->At_csr, this->y, this->Aty, 1.0, 0.0, this->SpMV_Aty_buffer);
            // hence Aty = A^T y^{k+1}

            // Rd1 <-- Aty
            CHECK_CUDA( hipMemcpy(this->Rd1.vals, this->Aty.vals, sizeof(double) * this->vec_len, D2D) );
            // Rd1 <-- (-1.0) * C + 1.0 * Rd1
            axpby_cusparse(this->cusparseH, this->C, this->Rd1, -1.0, 1.0);
            // hence Rd1 = A^T y^{k+1} - C
        }

        // when after the switch, use values computed in previous steps
        if (iter > this->switch_admm) {
            // if the current KKT residual is smaller than the best one so far,
            // update the best solution so far
            if (this->best_KKT > max(this->maxfeas, this->relgap)) {
                CHECK_CUDA( hipMemcpyAsync(this->X_best.vals, this->X.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[0].stream) );
                CHECK_CUDA( hipMemcpyAsync(this->y_best.vals, this->y.vals, sizeof(double) * this->con_num, D2D, this->stream_flex[1].stream) );
                CHECK_CUDA( hipMemcpyAsync(this->S_best.vals, this->S.vals, sizeof(double) * this->vec_len, D2D, this->stream_flex[2].stream) );
                this->best_KKT = max(this->maxfeas, this->relgap);
            }
        }


        /* Step 4: Compute X^{k+1} = X^k + tau * sigma (S^{k+1} + A^T y^{k+1} - C) */
        // Rd <-- 1.0 * Rd1 + 1.0 * S
        dense_vector_add_dense_vector(this->Rd, this->Rd1, this->S, 1.0, 1.0);
        if (iter < this->switch_admm) {
            this->tau = 1.95;
        } else {
            this->tau = 1.618; // (1 + sqrt(5)) / 2
        }
        if (this->errRd < stop_tol) {
            this->tau = max(1.618, this->tau / 1.1);
        }
        // hence Rd = Rd1 + S = A^T y^{k+1} - C + S

        // X <-- X + (tau * sig) * Rd
        dense_vector_add_dense_vector(this->X, this->Rd, 1.0, this->tau * this->sig);
        // hence X = X^k + (tau * sig) * (A^T y^{k+1} - C + S)

        /* Step "5": Compute KKT residuals, update parameters */

        // Rp <-- -1.0 * A * X + 0.0 * Rp
        SpMV_cusparse(this->cusparseH, this->A_csr, this->X, this->Rp, -1.0, 0.0, this->SpMV_AX_buffer);
        // hence Rp = - A X

        // Rp <-- 1.0 * b + 1.0 * Rp
        axpby_cusparse(this->cusparseH, this->b, this->Rp, 1.0, 1.0);
        // hence Rp = b - A X

        /* Update errors and compute residuals */
        dense_vector_mul_dense_vector_mul_scalar(this->Rporg, this->normA, this->Rp, this->bscale);
        this->errRp = this->Rporg.get_norm(this->cublasH) / this->norm_borg;
        this->pobj = SparseVV_cusparse(this->cusparseH, this->C, this->X, this->SpVV_CtX_buffer) * this->objscale;
        dense_vector_mul_scalar(this->Rdorg, this->Rd, this->Cscale);
        this->errRd = this->Rdorg.get_norm(this->cublasH) / this->norm_Corg;
        this->dobj = SparseVV_cusparse(this->cusparseH, this->b, this->y, this->SpVV_bty_buffer) * this->objscale;
        this->maxfeas = max(this->errRp, this->errRd);
        this->relgap = abs(this->pobj - this->dobj) / (1 + abs(this->pobj) + abs(this->dobj));
        this->feasratio = this->ratioconst * this->errRp / this->errRd;
        if (this->feasratio < 1) {
            this->prim_win += 1;
        } else {
            this->dual_win += 1;
        }

        /* Update sigma */
        if (
            ( (iter <= this->sig_update_threshold) && ((iter % this->sig_update_stage_1) == 1) ) ||
            ( (iter > this->sig_update_threshold) && ((iter % this->sig_update_stage_2) == 1) )
        ) {
            if (this->prim_win > 1.2 * this->dual_win) {
                this->prim_win = 0;
                this->sig = min(this->sigmax, this->sig * this->sigscale);
            } else if (this->dual_win > 1.2 * this->prim_win) {
                this->dual_win = 0;
                this->sig = max(this->sigmin, this->sig / this->sigscale);
            }
        }

        /* Add info */
        this->info_pobj_arr.push_back(this->pobj);
        this->info_dobj_arr.push_back(this->dobj);
        this->info_errRp_arr.push_back(this->errRp);
        this->info_errRd_arr.push_back(this->errRd);
        this->info_relgap_arr.push_back(this->relgap);
        this->info_sig_arr.push_back(this->sig);
        this->info_bscale_arr.push_back(this->bscale);
        this->info_Cscale_arr.push_back(this->Cscale);
        this->info_iter_num++;
    }

    // recover the original solution by unscaling
    dense_vector_mul_scalar(this->X, this->bscale);
    dense_vector_div_dense_vector_mul_scalar(this->y, this->normA, this->Cscale);
    dense_vector_mul_scalar(this->S, this->Cscale);

    // free the memory
    hipEventDestroy(this->start);
    hipEventDestroy(this->stop);

    return;
}