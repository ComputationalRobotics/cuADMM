#include "hip/hip_runtime.h"
/*

    dense_scalar.cu

    Defines CUDA kernel for operations between a dense vector and a scalar.

*/

#include "cuadmm/memory.h"

/* Kernels for coefficient-wise operations */

// Multiply in place the current coefficient of a dense vector by a scalar:
// vec[idx] *= scalar
__global__ void dense_vector_mul_scalar_kernel(
    double* vec_vals, int vec_size,
    double scalar
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vec_size) {
        vec_vals[idx] *= scalar; 
    }
    return;
}

// Multiply the current coefficient of a dense vector by a scalar and store the result in another vector:
// vec1[idx] = vec2[idx] * scalar
__global__ void dense_vector_mul_scalar_kernel(
    double* vec1_vals, double* vec2_vals, int vec_size,
    double scalar
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vec_size) {
        vec1_vals[idx] = vec2_vals[idx] * scalar; 
    }
    return;
}

// Set the current coefficient of a dense vector to its positive part:
// vec[idx] = max(vec[idx], 0)
__global__ void max_dense_vector_zero_kernel(double* vec_vals, int vec_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vec_size) {
        vec_vals[idx] = max(vec_vals[idx], 0.0);
    }
    return;
}

// Set the current coefficient of a dense vector to its positive part and multiply by a mask:
// vec[idx] = max(vec[idx], 0) * mask[idx]
__global__ void max_dense_vector_zero_mask_kernel(double* vec_vals, int* mask_vals, int vec_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vec_size) {
        vec_vals[idx] = max(vec_vals[idx], 0.0) * mask_vals[idx];
    }
    return;
}


/* Kernels for vector-wise operations */

// Multiply in place a vector by a scalar:
// vec *= scalar
void dense_vector_mul_scalar(DeviceDenseVector<double>& vec, double scalar, int block_size) {
    int num_block = (vec.size + block_size - 1) / block_size;
    dense_vector_mul_scalar_kernel<<<num_block, block_size>>>(vec.vals, vec.size, scalar);
    return;
}
void dense_vector_mul_scalar(DeviceDenseVector<double>& vec, double scalar, const hipStream_t& stream, int block_size) {
    int num_block = (vec.size + block_size - 1) / block_size;
    dense_vector_mul_scalar_kernel<<<num_block, block_size, 0, stream>>>(vec.vals, vec.size, scalar);
    return;
}

// Divide in place a vector by a scalar:
// vec *= scalar
void dense_vector_div_scalar(DeviceDenseVector<double>& vec, double scalar, const hipStream_t& stream, int block_size) {
    int num_block = (vec.size + block_size - 1) / block_size;
    dense_vector_mul_scalar_kernel<<<num_block, block_size, 0, stream>>>(vec.vals, vec.size, 1/scalar);
    return;
}

// Multiply a vector by a scalar and store the result in another vector:
// vec1 = vec2 * scalar
void dense_vector_mul_scalar(DeviceDenseVector<double>& vec1, DeviceDenseVector<double>& vec2, double scalar, const hipStream_t& stream, int block_size) {
    int num_block = (vec1.size + block_size - 1) / block_size;
    dense_vector_mul_scalar_kernel<<<num_block, block_size, 0, stream>>>(vec1.vals, vec2.vals, vec1.size, scalar);
    return;
}

// Set a vector to its positive part coefficient-wise:
// vec = max(vec, 0)
void max_dense_vector_zero(DeviceDenseVector<double>& vec, const hipStream_t& stream, int block_size) {
    int num_block = (vec.size + block_size - 1) / block_size;
    max_dense_vector_zero_kernel<<<num_block, block_size, 0, stream>>>(vec.vals, vec.size);
    return;
}

// Set a vector to its positive part coefficient-wise and multiply by a mask:
// vec = max(vec, 0) .* mask
void max_dense_vector_zero_mask(DeviceDenseVector<double>& vec, DeviceDenseVector<int>& mask, const hipStream_t& stream, int block_size) {
    int num_block = (vec.size + block_size - 1) / block_size;
    max_dense_vector_zero_mask_kernel<<<num_block, block_size, 0, stream>>>(vec.vals, mask.vals, vec.size);
    return;
}